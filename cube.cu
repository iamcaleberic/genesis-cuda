
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void cube(float * d_out, float * d_in){
  int idx = threadIdx.x;
  float f = d_in[idx];
  d_out[idx] = f * f * f;
}

int main(int argc, char ** argv){
  const int ARRAY_SIZE = 96;
  const int ARRAY_BYTES =  ARRAY_SIZE * sizeof(float);

  // generate input array on host
  // practice to declare host starting with h a and device with d
  float h_in[ARRAY_SIZE];
  for (int i = 0; i < ARRAY_SIZE; i++){
    h_in[i] = float(i);
  }
  float h_out[ARRAY_SIZE];

  // declare gpu memory pointers
  float * d_in;
  float * d_out;

  //allocate gpu memory

  hipMalloc((void **) &d_in, ARRAY_BYTES);
  hipMalloc((void **) &d_out, ARRAY_BYTES);

  // array transfering to gpu
  hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

  cube<<<1, ARRAY_SIZE>>>(d_out, d_in);

  // copy back to cpu
  hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

  for(int i = 0; i < ARRAY_SIZE; i++){
    printf("%f", h_out[i]);
    printf(((i % 4) != 3) ? "\t" : "\n");
  };

  // free memory
  hipFree(d_in);
  hipFree(d_out);

  return 0;
}
