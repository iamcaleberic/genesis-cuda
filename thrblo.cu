#include "hip/hip_runtime.h"
// using both threads and blocks
#include <stdlib.h>
#include <stdio.h>
#include "./random.h"

#define N (2048*2048)
#define THREADS_PER_BLOCK 512

__global__ void add(int *a, int *b, int *c) {
  int index = threadIdx.x + blockIdx.x * blockDim.x; //blockDim gets threads per block
  // if (index < n) // avoid accessing beyond array
    c[index] = a[index] + b[index];
}

int main(void) {
    int *a, *b ,*c;
    int *d_a, *d_b, *d_c ;
    int size = N * sizeof(int);

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Alloc space for host copies of a, b, c and setup input values
    a = (int *)malloc(size); random_ints(a, N);
    b = (int *)malloc(size); random_ints(b, N);
    c = (int *)malloc(size);

    // Copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Launch add() kernel on GPU with multiple blocks
    // add<<<(N + M-1) / M,M>>>(d_a, d_b, d_c, N);

    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Cleanup
    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    return 0;
}
