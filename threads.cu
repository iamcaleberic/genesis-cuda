#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include "./random.h"

#define N 512

__global__ void add(int *a, int *b, int *c) {
  c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

int main(void) {
    int *a, *b ,*c;
    int *d_a, *d_b, *d_c ;
    int size = N * sizeof(int);

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Alloc space for host copies of a, b, c and setup input values
    a = (int *)malloc(size); random_ints(a, N);
    b = (int *)malloc(size); random_ints(b, N);
    c = (int *)malloc(size);

    // Copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Launch add() kernel on GPU with multiple threads
    add<<<1,N>>>(d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Cleanup
    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    return 0;
}
